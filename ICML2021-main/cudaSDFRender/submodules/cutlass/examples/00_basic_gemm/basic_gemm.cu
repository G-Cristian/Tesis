#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>

// Helper methods to check for errors
#include "helper.h"

//
// CUTLASS includes needed for single-precision GEMM kernel
//

// Defines cutlass::gemm::device::Gemm, the generic Gemm computation template class.
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/epilogue/thread/linear_combination_relu.h"
#include "cutlass/gemm/device/default_gemm_configuration.h"

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassSgemmNN(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  // Define type definition for single-precision CUTLASS GEMM with column-major
  // input matrices and 128x128x8 threadblock tile size (chosen by default).
  using ColumnMajor = cutlass::layout::ColumnMajor;
  using ArchTag = cutlass::arch::Sm70;
  using OpClass = cutlass::arch::OpClassSimt;

  // fetch defaults since I'm not a gpu programmer!
  using DefaultConfig = cutlass::gemm::device::DefaultGemmConfiguration<OpClass, //op class
                                                                        ArchTag, // arch tag
                                                                        float,  // element a
                                                                        float,  // element b
                                                                        float,  // element c
                                                                        float>; // element accum

  // epilogues run within gemm kernel, once matmul complete.
  // this feature is main reason we are using cutlass
  //TODO: tanh....?  Why is only relu a specific op... very odd
  using ReLU_op = cutlass::epilogue::thread::LinearCombinationRelu<float,1>; // simt must operate on scalars!

  using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                  ColumnMajor,  // Layout of A matrix
                                                  float,        // Data-type of B matrix
                                                  ColumnMajor,  // Layout of B matrix
                                                  float,        // Data-type of C matrix
                                                  ColumnMajor,  // Layout of C matrix                                                             
                                                  float,        // Element Accumulator Type
                                                  OpClass,
                                                  ArchTag,
                                                  DefaultConfig::ThreadblockShape, 
                                                  DefaultConfig::WarpShape, 
                                                  DefaultConfig::InstructionShape,
                                                  ReLU_op>;


                                                                   
  // Define a CUTLASS GEMM type
  CutlassGemm gemm_operator;

  // Construct the CUTLASS GEMM arguments object.
  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source matrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta}); // Scalars used in the Epilogue


  // Launch the CUTLASS GEMM kernel.
  cutlass::cutStatus status = gemm_operator(args);

  // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  if (status != cutlass::cutStatus::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}

/// Kernel to initialize a matrix with small integers.
__global__ void InitializeMatrix_kernel(
  float *matrix,
  int ldm,
  int rows,
  int columns,
  int seed = 0) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < rows && j < columns) {
    int offset = i + j * ldm;

    // Generate arbitrary elements.
    int const k = 16807;
    int const m = 16;
    float value = float(((offset + seed) * k % m) - m / 2);

    matrix[offset] = value;
  }
}

/// Simple function to initialize a matrix to arbitrary small integers.
hipError_t InitializeMatrix(float *matrix, int ldm, int rows, int columns, int seed = 0) {

  dim3 block(16, 16);
  dim3 grid(
    (rows + block.x - 1) / block.x,
    (columns + block.y - 1) / block.y
  );

  InitializeMatrix_kernel<<< grid, block >>>(matrix, ldm, rows, columns, seed);

  return hipGetLastError();
}


/// Allocates device memory for a matrix then fills with arbitrary small integers.
hipError_t AllocateMatrix(float **matrix, int ldm, int rows, int columns, int seed = 0) {
  hipError_t result;

  size_t sizeof_matrix = sizeof(float) * ldm * columns;

  // Allocate device memory.
  result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to allocate matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Clear the allocation.
  result = hipMemset(*matrix, 0, sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to clear matrix device memory: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Initialize matrix elements to arbitrary small integers.
  result = InitializeMatrix(*matrix, ldm, rows, columns, seed);

  if (result != hipSuccess) {
    std::cerr << "Failed to initialize matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  return result;
}


/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS GEMM kernel.
hipError_t TestCutlassGemm(int M, int N, int K, float alpha, float beta) {
  hipError_t result;

  // Define several matrices to be used as operands to GEMM kernels.

  // Compute leading dimensions for each matrix.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // Compute size in bytes of the C matrix.
  size_t sizeof_C = sizeof(float) * ldc * N;

  // Define pointers to matrices in GPU device memory.
  float *A;
  float *B;
  float *C_cutlass;


  // Allocate matrices in GPU device memory with arbitrary seeds.
  result = AllocateMatrix(&A, lda, M, K, 0);

  if (result !=  hipSuccess) {
    return result;
  }

  result = AllocateMatrix(&B, ldb, K, N, 17);

  if (result !=  hipSuccess) {
    hipFree(A);
    return result;
  }

  result = AllocateMatrix(&C_cutlass, ldc, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    return result;
  }

  // Launch CUTLASS GEMM.
  result = CutlassSgemmNN(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc);
  result = CutlassSgemmNN(M, N, K, alpha, C_cutlass, ldc, B, ldb, beta, C_cutlass, ldc);  
  result = CutlassSgemmNN(M, N, K, alpha, C_cutlass, ldc, B, ldb, beta, C_cutlass, ldc);
  result = CutlassSgemmNN(M, N, K, alpha, C_cutlass, ldc, B, ldb, beta, C_cutlass, ldc);
  result = CutlassSgemmNN(M, N, K, alpha, C_cutlass, ldc, B, ldb, beta, C_cutlass, ldc);
  result = CutlassSgemmNN(M, N, K, alpha, C_cutlass, ldc, B, ldb, beta, C_cutlass, ldc);
  result = CutlassSgemmNN(M, N, K, alpha, C_cutlass, ldc, B, ldb, beta, C_cutlass, ldc);
  result = CutlassSgemmNN(M, N, K, alpha, C_cutlass, ldc, B, ldb, beta, C_cutlass, ldc);

  if (result != hipSuccess) {
    std::cerr << "CUTLASS GEMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  // Copy to host and verify equivalence.
  std::vector<float> host_cutlass(ldc * N, 0);

  result = hipMemcpy(host_cutlass.data(), C_cutlass, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy CUTLASS GEMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }



  //
  // Free device memory allocations.
  //

  hipFree(C_cutlass);
  hipFree(B);
  hipFree(A);

  return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Entry point to basic_gemm example.
//
// usage:
//
//   00_basic_gemm <M> <N> <K> <alpha> <beta>
//
int main(int argc, const char *arg[]) {

  //
  // Parse the command line to obtain GEMM dimensions and scalar values.
  //

  // GEMM problem dimensions.
  int problem[3] = { 16, 16, 16 };

  for (int i = 1; i < argc && i < 4; ++i) {
    std::stringstream ss(arg[i]);
    ss >> problem[i - 1];
  }

  // Scalars used for linear scaling the result of the matrix product.
  float scalars[2] = { 1, 1 };

  for (int i = 4; i < argc && i < 6; ++i) {
    std::stringstream ss(arg[i]);
    ss >> scalars[i - 4];
  }

  //
  // Run the CUTLASS GEMM test.
  //

  hipError_t result = TestCutlassGemm(
    problem[0],     // GEMM M dimension
    problem[1],     // GEMM N dimension
    problem[2],     // GEMM K dimension
    scalars[0],     // alpha
    scalars[1]      // beta
  );

  if (result == hipSuccess) {
    std::cout << "Passed." << std::endl;
  }

  // Exit.
  return result == hipSuccess ? 0 : -1;
}

